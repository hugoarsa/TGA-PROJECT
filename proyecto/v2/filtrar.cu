#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


#ifndef SIZE
#define SIZE 32
#endif

#ifndef PINNED
#define PINNED 0
#endif


// kernel elemento a elemento (no tiene pinta de que vaya a ser muy bueno)

__global__ void Gauss_kernel (int N, int M, unsigned char *source, unsigned char *dest) {

  __shared__ unsigned char s_source[SIZE][SIZE*3];

  float kernel[] = {0.0625, 0.125, 0.0625, 0.125, 0.25, 0.125, 0.0625, 0.125, 0.0625};
  int kernelSize = 3;
  int h_kernelSize = kernelSize / 2;

  int bx = blockIdx.x;  int sx = threadIdx.x;
  int by = blockIdx.y;  int sy = threadIdx.y;

  int i = by * blockDim.y + sy;
  int j = bx * blockDim.x + sx;

  //offsets para cargar shared
  int offset = i*M*3 + j*3;

  if (i < N && j < M) {
    //cada thread carga su pixel
    s_source[sy][sx*3] = source[offset];
    s_source[sy][sx*3+1] = source[offset+1];
    s_source[sy][sx*3+2] = source[offset+2];
    __syncthreads();

    float red = 0.0;
    float green = 0.0;
    float blue = 0.0;

    for (int k = 0; k < kernelSize; ++k){
      for (int l = 0; l < kernelSize; ++l){
        //indices para memoria principal
        int y = i + k - h_kernelSize;
        int x = (j + l - h_kernelSize)*3;
        //indices para shared
        int s_y = sy + k - h_kernelSize;
        int s_x = (sx + l - h_kernelSize)*3;

        if (y >= 0 && y < N && x >= 0 && x < M*3){ //dentro de principal
          if(s_y >= 0 && s_y < SIZE && s_x >= 0 && s_x < SIZE*3){ //dentro de shared
            red += s_source[s_y][s_x] * kernel[k*3 + l];
            green += s_source[s_y][s_x + 1] * kernel[k*3 + l];
            blue += s_source[s_y][s_x + 2] * kernel[k*3 + l];
          } else { //no lo hemos encontrado en shared y vamos a principal
            red += source[y*M*3 + x] * kernel[k*3 + l];
            green += source[y*M*3 + x + 1] * kernel[k*3 + l];
            blue += source[y*M*3 + x + 2] * kernel[k*3 + l];
          }
        }
      }
    }
    //pasamos los datos a destino
    dest[offset] = (unsigned char) red;
    dest[offset+1] = (unsigned char) green;
    dest[offset+2] = (unsigned char) blue;
  }
}


void InitM(int N, int M, float *Mat);


int main(int argc, char** argv)
{
  unsigned int numBytesI;
  unsigned int nBlocksW, nBlocksH, nThreads;

  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  unsigned char *h_output;
  unsigned char *d_image, *d_output;

  // Ficheros de entrada y de salida
  char *fileIN, *fileOUT;

  if (argc == 3) { fileIN = argv[1]; fileOUT = argv[2]; }
  else { printf("Usage: ./exe fileIN fileOUT\n"); exit(0); }


  // Lectura de imagenes
  unsigned char *h_image;
  //meta info de la imagen
  int width, height, pixelWidth;

  printf("Reading image...\n");
  h_image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);

  if (!h_image) {
    fprintf(stderr, "Couldn't load image.\n");
    return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);


  // Buscar GPU de forma aleatoria
  int count, gpu;
  hipGetDeviceCount(&count);
  srand(time(NULL));
  gpu = (rand()>>3) % count;
  hipSetDevice(gpu);


  // numero de Threads en cada dimension
  nThreads = SIZE;


  // numero de Blocks en cada dimension
  nBlocksW = (width+nThreads-1)/nThreads;
  nBlocksH = (height+nThreads-1)/nThreads;


  // calculamos la memoria a reservar
  numBytesI = width*height*3 * sizeof(unsigned char);


  // establecemos las dimensiones
  dim3 dimGrid(nBlocksW, nBlocksH, 1);
  dim3 dimBlock(nThreads, nThreads, 1);


  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);


  if (PINNED) {
    // Obtiene Memoria [pinned] en el host
    hipHostMalloc((unsigned char**)&h_output, numBytesI); 
  }
  else {
    // Obtener Memoria en el host
    h_output = (unsigned char*) malloc(numBytesI); 
  }

  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
  
  // Obtener Memoria en el device
  hipMalloc((unsigned char**)&d_image, numBytesI);
  hipMalloc((unsigned char**)&d_output, numBytesI);

  // Copiar datos desde el host en el device
  hipMemcpy(d_image, h_image, numBytesI, hipMemcpyHostToDevice);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);

  // Ejecutar el kernel
  Gauss_kernel<<<dimGrid, dimBlock>>>(height, width, d_image, d_output);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host
  hipMemcpy(h_output, d_output, numBytesI, hipMemcpyDeviceToHost);

  // Liberar Memoria del device
  hipFree(d_output);
  hipFree(d_image);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nKERNEL 01\n");
  printf("GPU utilizada: %d\n", gpu);
  printf("Dimensiones: height = %d, width = %d\n", height, width);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocksW, nBlocksH, nBlocksW*nBlocksH);
  if (PINNED) printf("Usando Pinned Memory\n");
         else printf("NO usa Pinned Memory\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
  //printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) M * (float) P) / (1000000.0 * TiempoTotal));
  //printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) M * (float) P) / (1000000.0 * TiempoKernel));

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,h_output,0);

  if (PINNED) {
    hipHostFree(h_output);
  }
  else {
    free(h_output);
  }

}
