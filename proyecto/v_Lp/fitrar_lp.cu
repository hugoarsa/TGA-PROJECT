#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


#ifndef SIZE
#define SIZE 32
#endif

#ifndef PINNED
#define PINNED 0
#endif


// kernel elemento a elemento para el filtro Laplace

__global__ void Laplace_kernel(int N, int M, unsigned char *source, unsigned char *dest) {
  int Kernel[] = {0, 1, 0, 1, -4, 1, 0, 1, 0};
  int kernelSize = 3;

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N && j < M) {
    int sum = 0;

    for (int k = 0; k < kernelSize; ++k) {
      for (int l = 0; l < kernelSize; ++l) {
        int y = i + k - kernelSize / 2;
        int x = j + l - kernelSize / 2;

        if (y >= 0 && y < N && x >= 0 && x < M) {
          sum += source[y * M + x] * Kernel[k * kernelSize + l];
        }
      }
    }

    int offset = i * M + j;
    dest[offset] = abs(sum);
  }
}


void InitM(int N, int M, float *Mat);


int main(int argc, char **argv) {
  unsigned int numBytesI;
  unsigned int nBlocksW, nBlocksH, nThreads;

  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  unsigned char *h_output;
  unsigned char *d_image, *d_output;

  // Ficheros de entrada y de salida
  char *fileIN, *fileOUT;

  if (argc == 3) {
    fileIN = argv[1];
    fileOUT = argv[2];
  } else {
    printf("Usage: ./exe fileIN fileOUT\n");
    exit(0);
  }


  // Lectura de imagenes
  unsigned char *h_image;
  // Meta info de la imagen
  int width, height, pixelWidth;

  printf("Reading image...\n");
  h_image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);

  if (!h_image) {
    fprintf(stderr, "Couldn't load image.\n");
    return (-1);
  }
  printf("Image Read. Width: %d, Height: %d, nComp: %d\n", width, height, pixelWidth);


  // Buscar GPU de forma aleatoria
  int count, gpu;
  hipGetDeviceCount(&count);
  srand(time(NULL));
  gpu = (rand() >> 3) % count;
  hipSetDevice(gpu);


  // Numero de Threads en cada dimension
  nThreads = SIZE;


  // Numero de Blocks en cada dimension
  nBlocksW = (width + nThreads - 1) / nThreads;
  nBlocksH = (height + nThreads - 1) / nThreads;


  // Calculamos la memoria a reservar
  numBytesI = width * height * 3 * sizeof(unsigned char);


  // Establecemos las dimensiones
  dim3 dimGrid(nBlocksW, nBlocksH, 1);
  dim3 dimBlock(nThreads, nThreads, 1);


  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);


  if (PINNED) {
    // Obtiene Memoria [pinned] en el host
    hipHostMalloc((unsigned char **)&h_output, numBytesI);
  } else {
    // Obtener Memoria en el host
    h_output = (unsigned char *)malloc(numBytesI);
  }

  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);

  // Obtener Memoria en el device
  hipMalloc((unsigned char **)&d_image, numBytesI);
  hipMalloc((unsigned char **)&d_output, numBytesI);

  // Copiar datos desde el host en el device
  hipMemcpy(d_image, h_image, numBytesI, hipMemcpyHostToDevice);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);

  // Ejecutar el kernel
  Laplace_kernel<<<dimGrid, dimBlock>>>(height, width, d_image, d_output);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host
  hipMemcpy(h_output, d_output, numBytesI, hipMemcpyDeviceToHost);

  // Liberar Memoria del device
  hipFree(d_output);
  hipFree(d_image);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal, E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nKERNEL 01\n");
  printf("GPU utilizada: %d\n", gpu);
  printf("Dimensiones: height = %d, width = %d\n", height, width);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocksW, nBlocksH, nBlocksW * nBlocksH);
  if (PINNED) printf("Usando Pinned Memory\n");
         else printf("NO usa Pinned Memory\n");
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
  //printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) M * (float) P) / (1000000.0 * TiempoTotal));
  //printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) M * (float) P) / (1000000.0 * TiempoKernel));

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,h_output,0);

  if (PINNED) {
    hipHostFree(h_output);
  }
  else {
    free(h_output);
  }

}